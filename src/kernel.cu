#include "hip/hip_runtime.h"
#include "kernel.h"

#include "pathtracing.cuh"
#include "view.h"
#include "world.h"
#include "camera.h"
#include ""
#include "cudaUtility.h"
#include "cutility.cuh"
#include "cutil_math.h"

#include <stdio.h>
#include <cstdlib>

float4* device_accum_image = NULL;

__host__
void kernelInit()
{
	hipMalloc(&device_accum_image, scene.camera.projection.num_pixels * sizeof(float4));
	hipMemset(device_accum_image, 0, scene.camera.projection.num_pixels * sizeof(float4));

	initPathTracing();
}

__host__
void kernelCleanUp()
{
	cleanUpPathTracing();
	hipFree(device_accum_image);
}



//Kernel that writes the image to the OpenGL PBO directly.
__global__
void writeImageToPBO(uchar4* pbo, float gamma, int width, int height, int iter, float4* image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < width && y < height) {
		int index = x + (y * width);

		float inv_iter = 1 / (float)iter;
		pbo[index].w = 0.0f;
		pbo[index].x = 255.0f * powf(clamp((image[index].x) * inv_iter, 0.0f, 1.0f), gamma);
		pbo[index].y = 255.0f * powf(clamp((image[index].y) * inv_iter, 0.0f, 1.0f), gamma);
		pbo[index].z = 255.0f * powf(clamp((image[index].z) * inv_iter, 0.0f, 1.0f), gamma);
	}
}


// Helper function for using CUDA to add vectors in parallel.
__host__
hipError_t kernelMain(uchar4* pbo, int iter)
{
	Camera& cam = scene.camera;
	// Launch a kernel on the GPU with one thread for each element.
	const int blockSideLength = 24;
	const dim3 blockSize(blockSideLength, blockSideLength);
	const dim3 blocksPerGrid(
		(cam.projection.width + blockSize.x - 1) / blockSize.x,
		(cam.projection.height + blockSize.y - 1) / blockSize.y);

	if (iter == 1) {
		// clear image
		hipMemset(device_accum_image, 0, cam.projection.num_pixels * sizeof(float4));
	}

	{ // pathtrace
		int iterHash = wang_hash(utilhash(iter));
		int jitterHash = wang_hash(iter);

		runPathTracing(iterHash, jitterHash);
		hipDeviceSynchronize();
		checkCudaError("run runPathTracing()");
	}

	// write results to pbo
	writeImageToPBO << <blocksPerGrid, blockSize >> >(	pbo,
														cam.projection.gamma_corr,
														cam.projection.width,
														cam.projection.height,
														iter,
														device_accum_image);
	checkCudaError("run sendImageToPBO<<<>>>()");
    
    hipDeviceSynchronize();
	checkCudaError("kernelMain<<< >>>()");

    return hipGetLastError();
}