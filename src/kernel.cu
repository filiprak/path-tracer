#include "hip/hip_runtime.h"

#include "view.h"
#include "hip/hip_runtime.h"
#include ""
#include "cudaUtility.h"

#include <stdio.h>
#include <cstdlib>
#include <thrust/random.h>
#include <glm\glm.hpp>

// temp image memory
static glm::vec3* dev_image = NULL;

__host__
void kernelCleanUp()
{
	hipFree(dev_image);
}

__host__
void kernelInit()
{
	int pixel_num = viewWidth * viewHeight;

	hipMalloc(&dev_image, pixel_num * sizeof(glm::vec3));
	hipMemset(dev_image, 0, pixel_num * sizeof(glm::vec3));
}

__host__ __device__
inline unsigned int utilhash(unsigned int a) {
	a = (a + 0x7ed55d16) + (a << 12);
	a = (a ^ 0xc761c23c) ^ (a >> 19);
	a = (a + 0x165667b1) + (a << 5);
	a = (a + 0xd3a2646c) ^ (a << 9);
	a = (a + 0xfd7046c5) + (a << 3);
	a = (a ^ 0xb55a4f09) ^ (a >> 16);
	return a;
}

__host__ __device__
thrust::default_random_engine makeSeededRandomEngine(int iter, int index, int depth) {
	int h = utilhash((1 << 31) | (depth << 22) | iter) ^ utilhash(index);
	return thrust::default_random_engine(h);
}

//Kernel that writes the image to the OpenGL PBO directly.
__global__
void writeImageToPBO(uchar4* pbo, int width, int height, int iter, glm::vec3* dev_image) {
	int x = (blockIdx.x * blockDim.x) + threadIdx.x;
	int y = (blockIdx.y * blockDim.y) + threadIdx.y;

	if (x < width && y < height) {
		int index = x + (y * width);

		thrust::default_random_engine rng = makeSeededRandomEngine(iter, index, 0);
		thrust::uniform_real_distribution<float> u01(0, 255.0);

		float r1 = u01(rng);
		float r2 = u01(rng);
		float r3 = u01(rng);

		dev_image[index] += glm::vec3(r1, r2, r3);

		// Each thread writes one pixel location in the texture (textel)
		pbo[index].w = 0;
		pbo[index].x = r1;
		pbo[index].y = r2;
		pbo[index].z = r3;

		if (x % 20 == 0 || y % 20 == 0) {
			pbo[index].x = 0.0;
			pbo[index].y = 0.0;
			pbo[index].z = 0.0;
		}
	}
}

// Helper function for using CUDA to add vectors in parallel.
__host__
hipError_t kernelMain(uchar4* pbo, int iter)
{
    // Launch a kernel on the GPU with one thread for each element.
	const int blockSideLength = 8;
	const dim3 blockSize(blockSideLength, blockSideLength);
	const dim3 blocksPerGrid(
		(viewWidth + blockSize.x - 1) / blockSize.x,
		(viewHeight + blockSize.y - 1) / blockSize.y);

	writeImageToPBO << <blocksPerGrid, blockSize >> >(pbo, viewWidth, viewHeight, iter, dev_image);
	checkCudaError("run sendImageToPBO<<< >>>()");
    
    hipDeviceSynchronize();
	checkCudaError("sendImageToPBO<<< >>>()");

    return hipGetLastError();
}
